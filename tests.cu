
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef TESTINGMODE

#include "gpuvm.h"
#include "testing.h"

#include "testing_vmstack.h"
#include "testing_vmheap.h"
#include "testing_vmengine.h"
#include "testing_opcodes.h"

int main()
{
    printf("Running tests\n\n");

    test_engine();
    test_heap();
    test_opcodes();
    test_stack();

    printf("Ran all tests\n\n");

    return 0;
}

#endif
