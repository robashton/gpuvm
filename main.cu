#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gpuvm.h"
#include "testing.h"


// TODO:
// Stack implementation                                                         [x]
// Heap implementation                                                          [x]
// Basic instruction processor                                                  [x]
// Method invocation                                                            [ ]
// Add primitive type 'float'                                                   [x]
// Add primitive type 'long'                                                    [ ]
// Add primitive type 'char'                                                    [ ]
// Add primitive type 'string'                                                  [ ]
// Add boot-strap code for initialising 'environment' (stack per thread etc)    [ ]
// Create wrapped malloc/free macros for use with GPU/CPU switch-over           [ ]
// Convert all functions to be __device__ and __global__ compatible             [ ]
// 'Actor' context (variable table pointing at global heap state)               [ ]
// Define a binary program 'format'                                             [ ]
// Write code to serialize/de-serialize into that format                        [ ]
// Define a DSL for actor based interaction                                     [ ]
// Create a compiler to compile DSL files                                       [ ]
// Create a linker to link DSL files into binary program format                 [ ]

/// THIS IS A PROBLEM FOR ANOTHER TIME
// Local variables? Global variables? How to avoid garbage collection?          [ ]

#ifndef TESTINGMODE

int main()
{
    printf("This will be the compiler/runtime utility/thingy, for now it's a blank space");
    return 0;
}

#endif
